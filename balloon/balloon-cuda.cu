#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/types.h>
//#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "balloon.h"
#include "../sha256-sse/sha256.h"
//#include <sys/time.h>

__global__ void conv_onethread(int n,int fn, const float * signal, const float * filter, float * retSignal);
__device__ void cuda_hash_state_mix (struct hash_state *s, int32_t mixrounds, uint64_t *prebuf_le);
__device__ void device_sha256_osol(const __sha256_block_t blk, __sha256_hash_t ctx);
__device__ void device_sha256_168byte(uint8_t *data, uint8_t *outhash);
__device__ void cuda_hash_state_extract (const struct hash_state *s, uint8_t out[BLOCK_SIZE]);
__device__ void cuda_compress (uint64_t *counter, uint8_t *out, const uint8_t *blocks[], size_t blocks_to_comp);
__device__ void cuda_expand (uint64_t *counter, uint8_t *buf, size_t blocks_in_buf);
__device__ void cuda_hash_state_fill (struct hash_state *s, const uint8_t *in, size_t inlen, int32_t t_cost, int64_t s_cost);
__device__ void device_sha256_generic(uint8_t *data, uint8_t *outhash, uint32_t len);
void host_sha256_osol(const __sha256_block_t blk, __sha256_hash_t ctx);
__global__ void cudaized_multi (struct hash_state *s, int32_t mixrounds, uint64_t *prebuf_le, uint8_t *input, uint32_t len, uint8_t *output, int64_t s_cost, uint32_t max_nonce, int gpuid, uint32_t *winning_nonce, uint32_t num_threads, uint32_t *device_target, uint32_t *is_winning, uint32_t num_blocks, uint8_t *sbufs);
void update_device_data(int gpuid);

//#define DEBUG
//#define CUDA_DEBUG
//#define CUDA_OUTPUT

//#define DEBUG
//#define DEBUG_CUDA
//#define LOWMEM

int cuda_query() {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
	}

	return nDevices;
}

__constant__ const uint32_t __sha256_init[] = {
    0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
    0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
};

#define PREBUF_LEN 409600
uint64_t host_prebuf_le[20][PREBUF_LEN / 8];
uint8_t host_prebuf_filled[20] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };
#define BLOCK_SIZE (32)

uint64_t *device_prebuf_le[20];
uint32_t *device_winning_nonce[20];
uint8_t *device_sbuf[20];
struct hash_state *device_s[20];
uint32_t *device_target[20];
uint32_t *device_is_winning[20];
uint8_t *device_out[20];
uint8_t *device_input[20];
uint8_t *device_sbufs[20];

uint8_t balloon_inited[20] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };
uint8_t syncmode_set[20] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };
void balloon_cuda_init(int gpuid, uint32_t opt_cuda_syncmode, uint32_t num_threads, uint32_t num_blocks) {
	checkCudaErrors(hipSetDevice(gpuid));
	if (!syncmode_set[gpuid]) {
		switch (opt_cuda_syncmode) {
		case 0:
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			break;
		case 1:
			hipSetDeviceFlags(hipDeviceScheduleSpin);
			break;
		case 2:
			hipSetDeviceFlags(hipDeviceScheduleYield);
			break;
		default:
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			break;
		}
		syncmode_set[gpuid] = 1;
	}
#ifdef DEBUG
	printf("DEBUG GPU %d: entering balloon_cuda_init\n", gpuid);
size_t free, total;
hipMemGetInfo(&free,&total); 
printf("%d KB free of total %d KB before init\n",free/1024,total/1024);
#endif
	if (!balloon_inited[gpuid]) {
		printf("Initiated GPU %d\n", gpuid);
		checkCudaErrors(hipMalloc((void**)&device_prebuf_le[gpuid], (PREBUF_LEN / 8) * sizeof(uint64_t)));
		checkCudaErrors(hipMalloc((void**)&device_sbuf[gpuid], /*s.n_blocks*/4096 * BLOCK_SIZE));
		checkCudaErrors(hipMalloc((void**)&device_is_winning[gpuid], sizeof(uint32_t)));
		checkCudaErrors(hipMalloc((void**)&device_winning_nonce[gpuid], sizeof(uint32_t)));
		checkCudaErrors(hipMalloc((void**)&device_s[gpuid], sizeof(struct hash_state)));
		checkCudaErrors(hipMalloc((void**)&device_target[gpuid], 8*sizeof(uint32_t)));
		checkCudaErrors(hipMalloc((void**)&device_out[gpuid], BLOCK_SIZE * sizeof(uint8_t)));
		checkCudaErrors(hipMalloc((void**)&device_input[gpuid], /*len*/80));
#ifdef LOWMEM
		checkCudaErrors(hipMalloc((void**)&device_sbufs[gpuid], num_threads*num_blocks*4096*BLOCK_SIZE));
		printf("device_sbufs[gpuid] = %x\n", device_sbufs[gpuid]);
#endif
		balloon_inited[gpuid] = 1;
	}
#ifdef DEBUG
	printf("DEBUG GPU %d: leaving balloon_cuda_init\n", gpuid);
hipMemGetInfo(&free,&total); 
printf("%d KB free of total %d KB after init\n",free/1024,total/1024);
#endif

}

void fill_prebuf(struct hash_state *s, int gpuid) {
#ifdef DEBUG
	printf("DEBUG GPU %d: entering fill_prebuf\n", gpuid);
#endif
	uint8_t host_prebuf[PREBUF_LEN];
	if (!host_prebuf_filled[gpuid]) {
		bitstream_fill_buffer (&s->bstream, host_prebuf, PREBUF_LEN);
		host_prebuf_filled[gpuid] = 1;
		uint8_t *buf = host_prebuf;
		uint64_t *lebuf = host_prebuf_le[gpuid];
		for (int i = 0; i < PREBUF_LEN; i+=8) {
			bytes_to_littleend8_uint64(buf, lebuf);
			*lebuf %= 4096;
			*lebuf <<= 5; // multiply by 32
			lebuf++;
			buf += 8;
		}
		update_device_data(gpuid);
		//printf("Filled prebuf for GPU %d\n", gpuid);
	}
#ifdef DEBUG
	printf("DEBUG GPU %d: leaving fill_prebuf\n", gpuid);
#endif
}

void reset_host_prebuf() {
	for (int i = 0; i < 20; i++) {
		host_prebuf_filled[i] = 0;
	}
}


void update_device_data(int gpuid) {
#ifdef DEBUG
	printf("DEBUG GPU %d: entering update_device_data\n", gpuid);
#endif
	checkCudaErrors(hipMemcpy(device_prebuf_le[gpuid], host_prebuf_le[gpuid], (PREBUF_LEN/8)*sizeof(uint64_t), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(device_prebuf_le), host_prebuf_le, 409600/8 * sizeof(uint64_t), 0, hipMemcpyHostToDevice));
#ifdef DEBUG
	printf("DEBUG GPU %d: leaving update_device_data\n", gpuid);
#endif
}

void balloon_cuda_free(int gpuid) {
	//hipFree(device_prebuf_le[gpuid]);
	//hipFree(device_sbuf[gpuid]);
	//hipFree(device_s[gpuid]);
	//hipFree(device_winning_nonce[gpuid]);
	//hipFree(device_is_winning[gpuid]);
	//hipFree(device_out[gpuid]);
	//hipFree(device_input[gpuid]);
#ifdef LOWMEM
	//hipFree(device_sbufs[gpuid]);
#endif
	//balloon_inited = 0;
}

uint32_t balloon_128_cuda (int gpuid, unsigned char *input, unsigned char *output, uint32_t *target, uint32_t max_nonce, uint32_t num_threads, uint32_t *is_winning, uint32_t num_blocks) {
	return cuda_balloon (gpuid, input, output, 80, 128, 4, target, max_nonce, num_threads, is_winning, num_blocks);
}

//#define NUM_THREADS 256
//#define NUM_THREADS 384
//#define NUM_THREADS 384
//#define NUM_BLOCKS 480
//#define NUM_BLOCKS 48


uint32_t cuda_balloon(int gpuid, unsigned char *input, unsigned char *output, int32_t len, int64_t s_cost, int32_t t_cost, uint32_t *target, uint32_t max_nonce, uint32_t num_threads, uint32_t *ret_is_winning, uint32_t num_blocks) {
#ifdef DEBUG
	printf("DEBUG GPU %d: entering cuda_balloon\n", gpuid);
#endif

	checkCudaErrors(hipSetDevice(gpuid));
	/*printf("input: ");
	for (int i = 0; i < len; i++) {
		printf("%02x ", input[i]);
	}
	printf("\n");*/
	struct balloon_options opts;
	struct hash_state s;
	balloon_init(&opts, s_cost, t_cost);
	hash_state_init(&s, &opts, input);
	fill_prebuf(&s, gpuid);
	uint8_t *pc_sbuf = s.buffer;

#ifdef DEBUG
	if (s.n_blocks > 4096) printf("s.n_blocks = %llu\n", s.n_blocks);
#endif

	uint32_t first_nonce = ((input[76] << 24) | (input[77] << 16) | (input[78] << 8) | input[79]);

	//printf("cuda_ballon, gpu %d, start_nonce: %d, max_nonce: %d\n", gpuid, first_nonce, max_nonce);

	checkCudaErrors(hipMemcpy((void**)device_sbuf[gpuid], (void**)s.buffer, s.n_blocks * BLOCK_SIZE, hipMemcpyHostToDevice));

	s.buffer = device_sbuf[gpuid];
	checkCudaErrors(hipMemcpy((void**)device_s[gpuid], (void**)&s, sizeof(struct hash_state), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy((void**)device_input[gpuid], (void**)input, len, hipMemcpyHostToDevice));
	uint32_t host_winning_nonce = 0;
	uint32_t host_is_winning = 0;

	checkCudaErrors(hipMemcpy(device_target[gpuid], target, 8 * sizeof(uint32_t), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy((void**)device_winning_nonce[gpuid], (void**)&host_winning_nonce, sizeof(uint32_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy((void**)device_is_winning[gpuid], (void**)&host_is_winning, sizeof(uint32_t), hipMemcpyHostToDevice));
	cudaized_multi << <num_blocks, num_threads >> > (device_s[gpuid], t_cost, device_prebuf_le[gpuid], device_input[gpuid], len, device_out[gpuid], s_cost, max_nonce, gpuid, device_winning_nonce[gpuid], num_threads, device_target[gpuid], device_is_winning[gpuid], num_blocks, device_sbufs[gpuid]);
	checkCudaErrors(hipPeekAtLastError());

	//wait for cuda device
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy((void*)&host_winning_nonce, (void*)device_winning_nonce[gpuid], sizeof(uint32_t), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((void*)&host_is_winning, (void*)device_is_winning[gpuid], sizeof(uint32_t), hipMemcpyDeviceToHost));
#ifdef DEBUG
	if (host_is_winning) {
		printf("[Host (GPU %d)] Winning (%d) nonce: %u\n", gpuid, host_is_winning, host_winning_nonce);
	}
#endif

#ifdef CUDA_OUTPUT
	checkCudaErrors(hipMemcpy((void**)output, (void**)device_out[gpuid], BLOCK_SIZE*sizeof(uint8_t), hipMemcpyDeviceToHost));
#endif



	s.buffer = pc_sbuf;
	hash_state_free(&s);

	*ret_is_winning = host_is_winning;
	if (host_is_winning == 0) {
		host_winning_nonce = first_nonce + num_threads*num_blocks - 1;
	}

	return host_winning_nonce;
}

__device__ void * block_index(const struct hash_state *s, size_t i) {
	return s->buffer + (BLOCK_SIZE * i);
}
__device__ void * block_last(const struct hash_state *s) {
	return block_index(s, s->n_blocks - 1);
}
__device__ void cuda_hash_state_extract(const struct hash_state *s, uint8_t out[BLOCK_SIZE]) {
	uint8_t *b = (uint8_t*)block_last(s);
	memcpy((char *)out, (const char *)b, BLOCK_SIZE);
}

//__device__ uint8_t local_input[NUM_BLOCKS*NUM_THREADS][80];
//__device__ uint8_t device_sbuf_init[4096*BLOCK_SIZE];
//__device__ struct hash_state local_s[NUM_BLOCKS*NUM_THREADS];
//__device__ uint8_t local_sbuf[NUM_BLOCKS*NUM_THREADS][4096*BLOCK_SIZE];
//__device__ uint32_t nonce[NUM_BLOCKS*NUM_THREADS];
//__device__ uint8_t local_output[NUM_BLOCKS*NUM_THREADS][32];

//#define CUDA_OUTPUT
__global__ void cudaized_multi(struct hash_state *hs, int32_t mixrounds, uint64_t *prebuf_le, uint8_t *input, uint32_t len, uint8_t *output, int64_t s_cost, uint32_t max_nonce, int gpuid, uint32_t *winning_nonce, uint32_t num_threads, uint32_t *device_target, uint32_t *is_winning, uint32_t num_blocks, uint8_t *sbufs) {
#ifdef DEBUG_CUDA
	printf("[Device %d] entering cuda\n", gpuid);
#endif
	uint32_t id = blockDim.x*blockIdx.x + threadIdx.x;
	uint32_t nonce = ((input[76] << 24) | (input[77] << 16) | (input[78] << 8) | input[79]) + id;
	if (nonce > max_nonce || *is_winning) {
#ifdef DEBUG_CUDA
		printf("[Device %d] winning_nonce flag already set, exiting\n", gpuid);
#endif
		asm("exit;");
	}
	uint8_t local_input[80];
#ifdef CUDA_OUTPUT
	uint8_t local_output[32];
#endif
	struct hash_state local_s;
	memcpy(local_input, input, len);
	memcpy(&local_s, hs, sizeof(struct hash_state));

#ifdef LOWMEM
	uint8_t *local_sbuf = sbufs+id*4096*BLOCK_SIZE;
#else
	uint8_t local_sbuf[4096*BLOCK_SIZE];
#endif

#ifdef LOWMEM
	memcpy(local_sbuf, hs->buffer, 4096 * BLOCK_SIZE);
#else
	memcpy(&local_sbuf, hs->buffer, 4096 * BLOCK_SIZE);
#endif

	local_s.buffer = local_sbuf;
	((uint32_t*)local_input)[19] = ((nonce & 0xff000000) >> 24) | ((nonce & 0xff0000) >> 8) | ((nonce & 0xff00) << 8) | ((nonce & 0xff) << 24);
	local_s.counter = 0;
	cuda_hash_state_fill(&local_s, local_input, len, mixrounds, s_cost);
	cuda_hash_state_mix (&local_s, mixrounds, prebuf_le);
#ifdef CUDA_OUTPUT
	cuda_hash_state_extract (&local_s, local_output);
	if (((uint32_t*)local_output)[7] < device_target[7]) {
#else
	if (((uint32_t*)(local_sbuf+(4095<<5)))[7] < device_target[7]) {
#endif
		// Assume winning nonce
#ifdef DEBUG
		printf("[Device %d] Winning nonce: %u\n", gpuid, nonce);
#endif
		*winning_nonce = nonce;
		*is_winning = 1;
#ifdef CUDA_OUTPUT
		memcpy(output, local_output, 32);
#endif
		__threadfence_system();
		asm("exit;");
	}
#ifdef DEBUG_CUDA
	printf("[Device %d] leaving cuda\n", gpuid);
#endif
}

__device__ void cuda_expand (uint64_t *counter, uint8_t *buf, size_t blocks_in_buf) {
  const uint8_t *blocks[1] = { buf };
  uint8_t *cur = buf + BLOCK_SIZE;
  for (size_t i = 1; i < blocks_in_buf; i++) {
    cuda_compress (counter, cur, blocks, 1);
    *blocks += BLOCK_SIZE;
    cur += BLOCK_SIZE;
  }
}

__device__ void cuda_compress (uint64_t *counter, uint8_t *out, const uint8_t *blocks[], size_t blocks_to_comp) {
	uint8_t data[168];
	uint8_t *dp = (uint8_t*)data;
	uint8_t len = BLOCK_SIZE * blocks_to_comp + 8;
	memcpy(dp, counter, 8);
	dp += 8;
	for (unsigned int i = 0; i < blocks_to_comp; i++) {
		memcpy(dp, *(blocks+i), BLOCK_SIZE);
		dp += BLOCK_SIZE;
	}
	device_sha256_generic(data, out, len);
	*counter += 1;
}

__device__ void cuda_hash_state_fill (struct hash_state *s, const uint8_t *in, size_t inlen, int32_t t_cost, int64_t s_cost) {
  uint8_t data[132];
  //uint32_t shalen = 8+SALT_LEN+inlen+8+4;
  uint8_t *dp = (uint8_t*)data;
  if (inlen != 80) {
	  printf("inlen != 128 (inlen = %d)!!\n", inlen);
	  if (inlen > 80) inlen = 80;
  }
  memcpy(dp, &s->counter, 8);
  dp += 8;
  memcpy(dp, in, SALT_LEN);
  dp += SALT_LEN;
  memcpy(dp, in, inlen);
  dp += inlen;
  memcpy(dp, &s_cost, 8);
  dp += 8;
  memcpy(dp, &t_cost, 4);

  device_sha256_generic(data, s->buffer, 132);
  s->counter++;
  cuda_expand (&s->counter, s->buffer, s->n_blocks);
}




__device__ void cuda_hash_state_mix (struct hash_state *s, int32_t mixrounds, uint64_t *prebuf_le) {
	uint64_t *buf = prebuf_le;
	uint8_t *sbuf = s->buffer;

	//int32_t n_blocks = s->n_blocks;
	const int32_t n_blocks = 4096;
	mixrounds = 4;
	uint8_t *last_block = (sbuf + (BLOCK_SIZE*(n_blocks-1)));
	uint8_t *blocks[5];
	unsigned char data[8 + BLOCK_SIZE * 5];
	unsigned char *db1 = data + 8;
	unsigned char *db2 = data + 40;
	unsigned char *db3 = data + 72;
	unsigned char *db4 = data + 104;
	unsigned char *db5 = data + 136;
	for (int32_t rounds=0; rounds < mixrounds; rounds++) {
		{ // i = 0
			blocks[0] = last_block;
			blocks[1] = sbuf;
			/*blocks[2] = (sbuf + (BLOCK_SIZE * (*(buf++))));
			blocks[3] = (sbuf + (BLOCK_SIZE * (*(buf++))));
			blocks[4] = (sbuf + (BLOCK_SIZE * (*(buf++))));*/

			blocks[2] = (sbuf + ((*(buf++))));
			blocks[3] = (sbuf + ((*(buf++))));
			blocks[4] = (sbuf + ((*(buf++))));

			// New sha256
			//block = (uint8_t**)blocks;
			memcpy(data, &s->counter, 8);
			memcpy(db1, blocks[0], BLOCK_SIZE);
			memcpy(db2, blocks[1], BLOCK_SIZE);
			memcpy(db3, blocks[2], BLOCK_SIZE);
			memcpy(db4, blocks[3], BLOCK_SIZE);
			memcpy(db5, blocks[4], BLOCK_SIZE);
			device_sha256_168byte(data, (uint8_t*)blocks[1]);
			s->counter++;
		}
		for (size_t i = 1; i < n_blocks; i++) {
			blocks[0] = blocks[1];
			blocks[1] += BLOCK_SIZE;
			/*blocks[2] = (sbuf + (BLOCK_SIZE * (*(buf++))));
			blocks[3] = (sbuf + (BLOCK_SIZE * (*(buf++))));
			blocks[4] = (sbuf + (BLOCK_SIZE * (*(buf++))));*/

			blocks[2] = (sbuf + ((*(buf++))));
			blocks[3] = (sbuf + ((*(buf++))));
			blocks[4] = (sbuf + ((*(buf++))));

			// New sha256
			memcpy(data, &s->counter, 8);
			memcpy(db1, blocks[0], BLOCK_SIZE);
			memcpy(db2, blocks[1], BLOCK_SIZE);
			memcpy(db3, blocks[2], BLOCK_SIZE);
			memcpy(db4, blocks[3], BLOCK_SIZE);
			memcpy(db5, blocks[4], BLOCK_SIZE);
			device_sha256_168byte(data, (uint8_t*)blocks[1]);
			s->counter++;
		}
		//s->has_mixed = true;
	}
#ifdef DEBUG_CUDA
	if (buf - prebuf_le > 49152) printf("prebuf_le max used: %d, mixrounds = %d, n_blocks = %d\n", buf - prebuf_le, mixrounds, n_blocks);
#endif
}

__device__ void device_sha256_168byte(uint8_t *data, uint8_t *outhash) {
	// outhash should be 32 byte
	//
	// l = 168byte => 1344bit (requires 3 blocks)
	// (k + 1 + l) mod 512 = 448
	// 512 * 3 = 1536 >= 1344:
	// k = 3*512 - 65 - l = 1536 - 65 - 1344 = 127 bits of padding => 15.875 bytes

	//__attribute__((aligned(16)))
	__sha256_block_t block[3];
	uint8_t *ptr = (uint8_t*)block;
	// 168 bytes of data
	memcpy(ptr, data, 168);
	ptr += 168;

	*ptr++ = 0x80; // End of string marker (and 7 bits padding)
	// Pad to (k+l+1 = 448 mod 512)
	// l = 168*8 = 1344bits
	// Blocks: 512bit | 512bit | 512bit
	// (512*3-65-l) = 1536-65-l = 1471 - l = 1471-1344 = 127bit = 15.875 bytes
	//memset(ptr, 0, 15);
	//ptr += 15;
	memset(ptr, 0, 21);
	ptr += 21;
	// 8 bytes is length (in bits)
	// 1344bit = 0x540
	/**ptr++ = 0x0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;*/
	*ptr++ = 0x5;
	*ptr++ = 0x40;

	__sha256_hash_t ohash;
	memcpy(ohash, __sha256_init, 32);
	device_sha256_osol(block[0], ohash);
	device_sha256_osol(block[1], ohash);
	device_sha256_osol(block[2], ohash);

	uint8_t *h = (uint8_t*)ohash;
	uint8_t *outp = outhash;
	for (int i = 0; i < 32/4; i++) {
		// Fix endianness at the same time
		*outp++ = h[3];
		*outp++ = h[2];
		*outp++ = h[1];
		*outp++ = h[0];
		h += 4;
	}
}

__device__ void device_sha256_generic(uint8_t *data, uint8_t *outhash, uint32_t len) {
	if (len > 184) {
		printf("Longer than 3 blocks (184bytes), sha256_generic not made for this..\n");
		len = 184;
	}
	uint8_t num_blocks = len/64 + 1;
	uint32_t tot_len = num_blocks*512 - 65; // 64bit header
	uint32_t num_padding = (tot_len - len*8)/8;

	//__attribute__((aligned(16)))
	__sha256_block_t block[3];
	uint8_t *ptr = (uint8_t*)block;
	memcpy(ptr, data, len);
	ptr += len;

	*ptr++ = 0x80; // End of string marker (and 7 bits padding)
	// Pad to (k+l+1 = 448 mod 512)
	// l = 168*8 = 1344bits
	// Blocks: 512bit | 512bit | 512bit
	// (512*3-65-l) = 1536-65-l = 1471 - l = 1471-1344 = 127bit = 15.875 bytes
	memset(ptr, 0, num_padding);
	ptr += num_padding;
	// 8 bytes is length (in bits)
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = ((len * 8) & 0xff00) >> 8;
	*ptr++ = (len * 8) & 0xff;

	__sha256_hash_t ohash;
	memcpy(ohash, __sha256_init, 32);
	for (int i = 0; i < num_blocks; i++) {
		device_sha256_osol(block[i], ohash);
	}

	uint8_t *h = (uint8_t*)ohash;
	uint8_t *outp = outhash;
	for (int i = 0; i < 32/4; i++) {
		// Fix endianness at the same time
		*outp++ = h[3];
		*outp++ = h[2];
		*outp++ = h[1];
		*outp++ = h[0];
		h += 4;
	}
}


/**************** SHA256 from github sha256-sse ***************/

#define	SHA256_CONST(x)		(SHA256_CONST_ ## x)

/* constants, as provided in FIPS 180-2 */

#define	SHA256_CONST_0		0x428a2f98U
#define	SHA256_CONST_1		0x71374491U
#define	SHA256_CONST_2		0xb5c0fbcfU
#define	SHA256_CONST_3		0xe9b5dba5U
#define	SHA256_CONST_4		0x3956c25bU
#define	SHA256_CONST_5		0x59f111f1U
#define	SHA256_CONST_6		0x923f82a4U
#define	SHA256_CONST_7		0xab1c5ed5U

#define	SHA256_CONST_8		0xd807aa98U
#define	SHA256_CONST_9		0x12835b01U
#define	SHA256_CONST_10		0x243185beU
#define	SHA256_CONST_11		0x550c7dc3U
#define	SHA256_CONST_12		0x72be5d74U
#define	SHA256_CONST_13		0x80deb1feU
#define	SHA256_CONST_14		0x9bdc06a7U
#define	SHA256_CONST_15		0xc19bf174U

#define	SHA256_CONST_16		0xe49b69c1U
#define	SHA256_CONST_17		0xefbe4786U
#define	SHA256_CONST_18		0x0fc19dc6U
#define	SHA256_CONST_19		0x240ca1ccU
#define	SHA256_CONST_20		0x2de92c6fU
#define	SHA256_CONST_21		0x4a7484aaU
#define	SHA256_CONST_22		0x5cb0a9dcU
#define	SHA256_CONST_23		0x76f988daU

#define	SHA256_CONST_24		0x983e5152U
#define	SHA256_CONST_25		0xa831c66dU
#define	SHA256_CONST_26		0xb00327c8U
#define	SHA256_CONST_27		0xbf597fc7U
#define	SHA256_CONST_28		0xc6e00bf3U
#define	SHA256_CONST_29		0xd5a79147U
#define	SHA256_CONST_30		0x06ca6351U
#define	SHA256_CONST_31		0x14292967U

#define	SHA256_CONST_32		0x27b70a85U
#define	SHA256_CONST_33		0x2e1b2138U
#define	SHA256_CONST_34		0x4d2c6dfcU
#define	SHA256_CONST_35		0x53380d13U
#define	SHA256_CONST_36		0x650a7354U
#define	SHA256_CONST_37		0x766a0abbU
#define	SHA256_CONST_38		0x81c2c92eU
#define	SHA256_CONST_39		0x92722c85U

#define	SHA256_CONST_40		0xa2bfe8a1U
#define	SHA256_CONST_41		0xa81a664bU
#define	SHA256_CONST_42		0xc24b8b70U
#define	SHA256_CONST_43		0xc76c51a3U
#define	SHA256_CONST_44		0xd192e819U
#define	SHA256_CONST_45		0xd6990624U
#define	SHA256_CONST_46		0xf40e3585U
#define	SHA256_CONST_47		0x106aa070U

#define	SHA256_CONST_48		0x19a4c116U
#define	SHA256_CONST_49		0x1e376c08U
#define	SHA256_CONST_50		0x2748774cU
#define	SHA256_CONST_51		0x34b0bcb5U
#define	SHA256_CONST_52		0x391c0cb3U
#define	SHA256_CONST_53		0x4ed8aa4aU
#define	SHA256_CONST_54		0x5b9cca4fU
#define	SHA256_CONST_55		0x682e6ff3U

#define	SHA256_CONST_56		0x748f82eeU
#define	SHA256_CONST_57		0x78a5636fU
#define	SHA256_CONST_58		0x84c87814U
#define	SHA256_CONST_59		0x8cc70208U
#define	SHA256_CONST_60		0x90befffaU
#define	SHA256_CONST_61		0xa4506cebU
#define	SHA256_CONST_62		0xbef9a3f7U
#define	SHA256_CONST_63		0xc67178f2U

/* Ch and Maj are the basic SHA2 functions. */
#define	Ch(b, c, d)	(((b) & (c)) ^ ((~b) & (d)))
#define	Maj(b, c, d)	(((b) & (c)) ^ ((b) & (d)) ^ ((c) & (d)))

/* Rotates x right n bits. */
#define	ROTR(x, n)	\
(((x) >> (n)) | ((x) << ((sizeof (x) * 8)-(n))))

/* Shift x right n bits */
#define	SHR(x, n)	((x) >> (n))

/* SHA256 Functions */
#define	BIGSIGMA0_256(x)	(ROTR((x), 2) ^ ROTR((x), 13) ^ ROTR((x), 22))
#define	BIGSIGMA1_256(x)	(ROTR((x), 6) ^ ROTR((x), 11) ^ ROTR((x), 25))
#define	SIGMA0_256(x)		(ROTR((x), 7) ^ ROTR((x), 18) ^ SHR((x), 3))
#define	SIGMA1_256(x)		(ROTR((x), 17) ^ ROTR((x), 19) ^ SHR((x), 10))

#define	SHA256ROUND(a, b, c, d, e, f, g, h, i, w)			\
T1 = h + BIGSIGMA1_256(e) + Ch(e, f, g) + SHA256_CONST(i) + w;	\
d += T1;							\
T2 = BIGSIGMA0_256(a) + Maj(a, b, c);				\
h = T1 + T2


/*
 * sparc optimization:
 *
 * on the sparc, we can load big endian 32-bit data easily.  note that
 * special care must be taken to ensure the address is 32-bit aligned.
 * in the interest of speed, we don't check to make sure, since
 * careful programming can guarantee this for us.
 */

#if	defined(_BIG_ENDIAN)
#define	LOAD_BIG_32(addr)	(*(uint32_t *)(addr))
#define	LOAD_BIG_64(addr)	(*(uint64_t *)(addr))

#elif	defined(HAVE_HTONL)
#define	LOAD_BIG_32(addr) htonl(*((uint32_t *)(addr)))
#define	LOAD_BIG_64(addr) htonll(*((uint64_t *)(addr)))

#else
/* little endian -- will work on big endian, but slowly */
#define	LOAD_BIG_32(addr)	\
(((addr)[0] << 24) | ((addr)[1] << 16) | ((addr)[2] << 8) | (addr)[3])
#define	LOAD_BIG_64(addr)	\
(((uint64_t)(addr)[0] << 56) | ((uint64_t)(addr)[1] << 48) |	\
((uint64_t)(addr)[2] << 40) | ((uint64_t)(addr)[3] << 32) |	\
((uint64_t)(addr)[4] << 24) | ((uint64_t)(addr)[5] << 16) |	\
((uint64_t)(addr)[6] << 8) | (uint64_t)(addr)[7])
#endif	/* _BIG_ENDIAN */

#if 0
#define dumpstate() printf("%s: %08x %08x %08x %08x %08x %08x %08x %08x %08x\n", __func__, w0, a, b, c, d, e, f, g, h);
#else
#define dumpstate()
#endif
void host_sha256_osol(const __sha256_block_t blk, __sha256_hash_t ctx) {
	uint32_t a = ctx[0];
	uint32_t b = ctx[1];
	uint32_t c = ctx[2];
	uint32_t d = ctx[3];
	uint32_t e = ctx[4];
	uint32_t f = ctx[5];
	uint32_t g = ctx[6];
	uint32_t h = ctx[7];

	uint32_t w0, w1, w2, w3, w4, w5, w6, w7;
	uint32_t w8, w9, w10, w11, w12, w13, w14, w15;
	uint32_t T1, T2;

	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w0 =  LOAD_BIG_32(blk + 4 * 0);
    dumpstate();
	SHA256ROUND(a, b, c, d, e, f, g, h, 0, w0);
    dumpstate();

	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w1 =  LOAD_BIG_32(blk + 4 * 1);
	SHA256ROUND(h, a, b, c, d, e, f, g, 1, w1);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w2 =  LOAD_BIG_32(blk + 4 * 2);
	SHA256ROUND(g, h, a, b, c, d, e, f, 2, w2);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w3 =  LOAD_BIG_32(blk + 4 * 3);
	SHA256ROUND(f, g, h, a, b, c, d, e, 3, w3);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w4 =  LOAD_BIG_32(blk + 4 * 4);
	SHA256ROUND(e, f, g, h, a, b, c, d, 4, w4);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w5 =  LOAD_BIG_32(blk + 4 * 5);
	SHA256ROUND(d, e, f, g, h, a, b, c, 5, w5);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w6 =  LOAD_BIG_32(blk + 4 * 6);
	SHA256ROUND(c, d, e, f, g, h, a, b, 6, w6);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w7 =  LOAD_BIG_32(blk + 4 * 7);
	SHA256ROUND(b, c, d, e, f, g, h, a, 7, w7);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w8 =  LOAD_BIG_32(blk + 4 * 8);
	SHA256ROUND(a, b, c, d, e, f, g, h, 8, w8);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w9 =  LOAD_BIG_32(blk + 4 * 9);
	SHA256ROUND(h, a, b, c, d, e, f, g, 9, w9);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w10 =  LOAD_BIG_32(blk + 4 * 10);
	SHA256ROUND(g, h, a, b, c, d, e, f, 10, w10);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w11 =  LOAD_BIG_32(blk + 4 * 11);
	SHA256ROUND(f, g, h, a, b, c, d, e, 11, w11);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w12 =  LOAD_BIG_32(blk + 4 * 12);
	SHA256ROUND(e, f, g, h, a, b, c, d, 12, w12);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w13 =  LOAD_BIG_32(blk + 4 * 13);
	SHA256ROUND(d, e, f, g, h, a, b, c, 13, w13);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w14 =  LOAD_BIG_32(blk + 4 * 14);
	SHA256ROUND(c, d, e, f, g, h, a, b, 14, w14);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w15 =  LOAD_BIG_32(blk + 4 * 15);
	SHA256ROUND(b, c, d, e, f, g, h, a, 15, w15);

	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0;
	SHA256ROUND(a, b, c, d, e, f, g, h, 16, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1;
	SHA256ROUND(h, a, b, c, d, e, f, g, 17, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2;
	SHA256ROUND(g, h, a, b, c, d, e, f, 18, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3;
	SHA256ROUND(f, g, h, a, b, c, d, e, 19, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4;
	SHA256ROUND(e, f, g, h, a, b, c, d, 20, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5;
	SHA256ROUND(d, e, f, g, h, a, b, c, 21, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6;
	SHA256ROUND(c, d, e, f, g, h, a, b, 22, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7;
	SHA256ROUND(b, c, d, e, f, g, h, a, 23, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8;
	SHA256ROUND(a, b, c, d, e, f, g, h, 24, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9;
	SHA256ROUND(h, a, b, c, d, e, f, g, 25, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10;
	SHA256ROUND(g, h, a, b, c, d, e, f, 26, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11;
	SHA256ROUND(f, g, h, a, b, c, d, e, 27, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12;
	SHA256ROUND(e, f, g, h, a, b, c, d, 28, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13;
	SHA256ROUND(d, e, f, g, h, a, b, c, 29, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14;
	SHA256ROUND(c, d, e, f, g, h, a, b, 30, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15;
	SHA256ROUND(b, c, d, e, f, g, h, a, 31, w15);

	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0;
	SHA256ROUND(a, b, c, d, e, f, g, h, 32, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1;
	SHA256ROUND(h, a, b, c, d, e, f, g, 33, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2;
	SHA256ROUND(g, h, a, b, c, d, e, f, 34, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3;
	SHA256ROUND(f, g, h, a, b, c, d, e, 35, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4;
	SHA256ROUND(e, f, g, h, a, b, c, d, 36, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5;
	SHA256ROUND(d, e, f, g, h, a, b, c, 37, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6;
	SHA256ROUND(c, d, e, f, g, h, a, b, 38, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7;
	SHA256ROUND(b, c, d, e, f, g, h, a, 39, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8;
	SHA256ROUND(a, b, c, d, e, f, g, h, 40, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9;
	SHA256ROUND(h, a, b, c, d, e, f, g, 41, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10;
	SHA256ROUND(g, h, a, b, c, d, e, f, 42, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11;
	SHA256ROUND(f, g, h, a, b, c, d, e, 43, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12;
	SHA256ROUND(e, f, g, h, a, b, c, d, 44, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13;
	SHA256ROUND(d, e, f, g, h, a, b, c, 45, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14;
	SHA256ROUND(c, d, e, f, g, h, a, b, 46, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15;
	SHA256ROUND(b, c, d, e, f, g, h, a, 47, w15);

	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0;
	SHA256ROUND(a, b, c, d, e, f, g, h, 48, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1;
	SHA256ROUND(h, a, b, c, d, e, f, g, 49, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2;
	SHA256ROUND(g, h, a, b, c, d, e, f, 50, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3;
	SHA256ROUND(f, g, h, a, b, c, d, e, 51, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4;
	SHA256ROUND(e, f, g, h, a, b, c, d, 52, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5;
	SHA256ROUND(d, e, f, g, h, a, b, c, 53, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6;
	SHA256ROUND(c, d, e, f, g, h, a, b, 54, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7;
	SHA256ROUND(b, c, d, e, f, g, h, a, 55, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8;
	SHA256ROUND(a, b, c, d, e, f, g, h, 56, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9;
	SHA256ROUND(h, a, b, c, d, e, f, g, 57, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10;
	SHA256ROUND(g, h, a, b, c, d, e, f, 58, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11;
	SHA256ROUND(f, g, h, a, b, c, d, e, 59, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12;
	SHA256ROUND(e, f, g, h, a, b, c, d, 60, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13;
	SHA256ROUND(d, e, f, g, h, a, b, c, 61, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14;
	SHA256ROUND(c, d, e, f, g, h, a, b, 62, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15;
	SHA256ROUND(b, c, d, e, f, g, h, a, 63, w15);

    //printf("%s last d: %08x\n", __func__, d);

    //printf("%s a: %08x %08x\n", __func__, a, ctx[0]);
	ctx[0] += a;
    //printf("%s a: %08x\n", __func__, ctx[0]);
	ctx[1] += b;
	ctx[2] += c;
	ctx[3] += d;
	ctx[4] += e;
	ctx[5] += f;
	ctx[6] += g;
	ctx[7] += h;

}

__device__ void device_sha256_osol(const __sha256_block_t blk, __sha256_hash_t ctx) {
	uint32_t a = ctx[0];
	uint32_t b = ctx[1];
	uint32_t c = ctx[2];
	uint32_t d = ctx[3];
	uint32_t e = ctx[4];
	uint32_t f = ctx[5];
	uint32_t g = ctx[6];
	uint32_t h = ctx[7];

	uint32_t w0, w1, w2, w3, w4, w5, w6, w7;
	uint32_t w8, w9, w10, w11, w12, w13, w14, w15;
	uint32_t T1, T2;

	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w0 =  LOAD_BIG_32(blk + 4 * 0);
    dumpstate();
	SHA256ROUND(a, b, c, d, e, f, g, h, 0, w0);
    dumpstate();

	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w1 =  LOAD_BIG_32(blk + 4 * 1);
	SHA256ROUND(h, a, b, c, d, e, f, g, 1, w1);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w2 =  LOAD_BIG_32(blk + 4 * 2);
	SHA256ROUND(g, h, a, b, c, d, e, f, 2, w2);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w3 =  LOAD_BIG_32(blk + 4 * 3);
	SHA256ROUND(f, g, h, a, b, c, d, e, 3, w3);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w4 =  LOAD_BIG_32(blk + 4 * 4);
	SHA256ROUND(e, f, g, h, a, b, c, d, 4, w4);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w5 =  LOAD_BIG_32(blk + 4 * 5);
	SHA256ROUND(d, e, f, g, h, a, b, c, 5, w5);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w6 =  LOAD_BIG_32(blk + 4 * 6);
	SHA256ROUND(c, d, e, f, g, h, a, b, 6, w6);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w7 =  LOAD_BIG_32(blk + 4 * 7);
	SHA256ROUND(b, c, d, e, f, g, h, a, 7, w7);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w8 =  LOAD_BIG_32(blk + 4 * 8);
	SHA256ROUND(a, b, c, d, e, f, g, h, 8, w8);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w9 =  LOAD_BIG_32(blk + 4 * 9);
	SHA256ROUND(h, a, b, c, d, e, f, g, 9, w9);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w10 =  LOAD_BIG_32(blk + 4 * 10);
	SHA256ROUND(g, h, a, b, c, d, e, f, 10, w10);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w11 =  LOAD_BIG_32(blk + 4 * 11);
	SHA256ROUND(f, g, h, a, b, c, d, e, 11, w11);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w12 =  LOAD_BIG_32(blk + 4 * 12);
	SHA256ROUND(e, f, g, h, a, b, c, d, 12, w12);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w13 =  LOAD_BIG_32(blk + 4 * 13);
	SHA256ROUND(d, e, f, g, h, a, b, c, 13, w13);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w14 =  LOAD_BIG_32(blk + 4 * 14);
	SHA256ROUND(c, d, e, f, g, h, a, b, 14, w14);
	/* LINTED E_BAD_PTR_CAST_ALIGN */
	w15 =  LOAD_BIG_32(blk + 4 * 15);
	SHA256ROUND(b, c, d, e, f, g, h, a, 15, w15);

	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0;
	SHA256ROUND(a, b, c, d, e, f, g, h, 16, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1;
	SHA256ROUND(h, a, b, c, d, e, f, g, 17, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2;
	SHA256ROUND(g, h, a, b, c, d, e, f, 18, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3;
	SHA256ROUND(f, g, h, a, b, c, d, e, 19, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4;
	SHA256ROUND(e, f, g, h, a, b, c, d, 20, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5;
	SHA256ROUND(d, e, f, g, h, a, b, c, 21, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6;
	SHA256ROUND(c, d, e, f, g, h, a, b, 22, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7;
	SHA256ROUND(b, c, d, e, f, g, h, a, 23, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8;
	SHA256ROUND(a, b, c, d, e, f, g, h, 24, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9;
	SHA256ROUND(h, a, b, c, d, e, f, g, 25, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10;
	SHA256ROUND(g, h, a, b, c, d, e, f, 26, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11;
	SHA256ROUND(f, g, h, a, b, c, d, e, 27, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12;
	SHA256ROUND(e, f, g, h, a, b, c, d, 28, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13;
	SHA256ROUND(d, e, f, g, h, a, b, c, 29, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14;
	SHA256ROUND(c, d, e, f, g, h, a, b, 30, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15;
	SHA256ROUND(b, c, d, e, f, g, h, a, 31, w15);

	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0;
	SHA256ROUND(a, b, c, d, e, f, g, h, 32, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1;
	SHA256ROUND(h, a, b, c, d, e, f, g, 33, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2;
	SHA256ROUND(g, h, a, b, c, d, e, f, 34, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3;
	SHA256ROUND(f, g, h, a, b, c, d, e, 35, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4;
	SHA256ROUND(e, f, g, h, a, b, c, d, 36, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5;
	SHA256ROUND(d, e, f, g, h, a, b, c, 37, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6;
	SHA256ROUND(c, d, e, f, g, h, a, b, 38, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7;
	SHA256ROUND(b, c, d, e, f, g, h, a, 39, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8;
	SHA256ROUND(a, b, c, d, e, f, g, h, 40, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9;
	SHA256ROUND(h, a, b, c, d, e, f, g, 41, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10;
	SHA256ROUND(g, h, a, b, c, d, e, f, 42, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11;
	SHA256ROUND(f, g, h, a, b, c, d, e, 43, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12;
	SHA256ROUND(e, f, g, h, a, b, c, d, 44, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13;
	SHA256ROUND(d, e, f, g, h, a, b, c, 45, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14;
	SHA256ROUND(c, d, e, f, g, h, a, b, 46, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15;
	SHA256ROUND(b, c, d, e, f, g, h, a, 47, w15);

	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0;
	SHA256ROUND(a, b, c, d, e, f, g, h, 48, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1;
	SHA256ROUND(h, a, b, c, d, e, f, g, 49, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2;
	SHA256ROUND(g, h, a, b, c, d, e, f, 50, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3;
	SHA256ROUND(f, g, h, a, b, c, d, e, 51, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4;
	SHA256ROUND(e, f, g, h, a, b, c, d, 52, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5;
	SHA256ROUND(d, e, f, g, h, a, b, c, 53, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6;
	SHA256ROUND(c, d, e, f, g, h, a, b, 54, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7;
	SHA256ROUND(b, c, d, e, f, g, h, a, 55, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8;
	SHA256ROUND(a, b, c, d, e, f, g, h, 56, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9;
	SHA256ROUND(h, a, b, c, d, e, f, g, 57, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10;
	SHA256ROUND(g, h, a, b, c, d, e, f, 58, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11;
	SHA256ROUND(f, g, h, a, b, c, d, e, 59, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12;
	SHA256ROUND(e, f, g, h, a, b, c, d, 60, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13;
	SHA256ROUND(d, e, f, g, h, a, b, c, 61, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14;
	SHA256ROUND(c, d, e, f, g, h, a, b, 62, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15;
	SHA256ROUND(b, c, d, e, f, g, h, a, 63, w15);

    //printf("%s last d: %08x\n", __func__, d);

    //printf("%s a: %08x %08x\n", __func__, a, ctx[0]);
	ctx[0] += a;
    //printf("%s a: %08x\n", __func__, ctx[0]);
	ctx[1] += b;
	ctx[2] += c;
	ctx[3] += d;
	ctx[4] += e;
	ctx[5] += f;
	ctx[6] += g;
	ctx[7] += h;

}
